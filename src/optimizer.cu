#include "hip/hip_runtime.h"
#include "header.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void sgd_kernel(float* param, const float* grad, float learning_rate, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        param[idx] -= learning_rate * grad[idx];
    }
}

void sgd(float* param, const float* grad, float learning_rate, int n, int threads_per_block) {
    // 디바이스 메모리 할당
    float *d_param, *d_grad;
    hipMalloc(&d_param, n * sizeof(float));
    hipMalloc(&d_grad, n * sizeof(float));
    
    // 호스트 → 디바이스 복사
    hipMemcpy(d_param, param, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_grad, grad, n * sizeof(float), hipMemcpyHostToDevice);
    
    // 커널 실행
    int blocks = (n + threads_per_block - 1) / threads_per_block;
    sgd_kernel<<<blocks, threads_per_block>>>(d_param, d_grad, learning_rate, n);
    
    // 디바이스 → 호스트 복사 (업데이트된 파라미터)
    hipMemcpy(param, d_param, n * sizeof(float), hipMemcpyDeviceToHost);
    
    // 메모리 해제
    hipFree(d_param);
    hipFree(d_grad);
    
    // 오류 확인
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA SGD error: %s\n", hipGetErrorString(err));
    }
}