#include "hip/hip_runtime.h"
#include "header.cuh"

__global__ void relu_kernel(float* input, float* output, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        output[idx] = fmaxf(0.0f, input[idx]);
    }
}

void relu(const float* input, float* output, int n, int threads_per_block) {
    float* d_input;
    float* d_output;

    hipMalloc(&d_input, n * sizeof(float));
    hipMalloc(&d_output, n * sizeof(float));

    hipMemcpy(d_input, input, n * sizeof(float), hipMemcpyHostToDevice);

    int blocks = (n + threads_per_block - 1) / threads_per_block;
    relu_kernel<<<blocks, threads_per_block>>>(d_input, d_output, n);

    hipMemcpy(output, d_output, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
